#include "hip/hip_runtime.h"
/* This software contains source code provided by NVIDIA Corporation. */

#include <iostream>
#include "mpigpu.h"

extern void abort(int);


__global__ void kernel(real* input, real* output) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  output[tid] = sqrt(input[tid]);
}

void compute(real* hostData, int blockSize, int gridSize) {
  int dataSize = blockSize * gridSize;
  int memsize  = dataSize * sizeof(real);

  // Allocate data on GPU memory
  real* deviceInputData = NULL;
  hipMalloc((void**)&deviceInputData, memsize);

  real* deviceOutputData = NULL;
  hipMalloc((void**)&deviceOutputData, memsize);

  // copy to GPU
  hipMemcpy(deviceInputData, hostData, memsize, hipMemcpyHostToDevice);

  // Run kernel
  kernel<<<gridSize, blockSize>>>(deviceInputData, deviceOutputData);

  // Copy to CPU
  hipMemcpy(hostData, deviceOutputData, memsize, hipMemcpyDeviceToHost);

  hipFree(deviceInputData);
  hipFree(deviceOutputData);
}
